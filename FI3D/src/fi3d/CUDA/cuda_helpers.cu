#ifndef FI3D_NO_CUDA

//#include <fi3d/Logger/Logger.h>

#include <fi3d/CUDA/cuda_helpers.h>
#include <fi3d/CUDA/cuda_computations.cuh>

#include <hip/hip_runtime.h>
#include <>

using namespace fi3d;
using namespace fi3d::CUDA;

/// Checks whether the was an error using any CUDA functionality. 
/// Call this method after every CUDA call.
#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__);}
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = false) {
	if (code != hipSuccess) {
		if (abort) {
			//LOG(fatal) << hipGetErrorString(code);
			exit(code);
		}
		//LOG(error) << hipGetErrorString(code);
	}
}

void computeMK3Workspace_sphere(ModelData& workspaceModel, float separation, 
	float depth, float range, int translations, int rotations) 
{
	//if (translations > 1024) {
	//	translations = 1024;
	//}
	//if (rotations > 1024) {
	//	rotations = 1024;
	//}

	//float szTranslation = range / (float)translations;		
	//float szRotation = 6.28318f / (float)rotations;
	//int totalPoints = translations * rotations;

	//float* rawPoints = 0;
	//gpuErrorCheck(hipMallocManaged(&rawPoints, totalPoints * workspaceModel.getPointSize()));
	//gpuErrorCheck(hipMemset(rawPoints, 0, totalPoints * workspaceModel.getPointSize()));
	//computeMK3Workspace_sphere_cuda<<<translations, rotations>>>(rawPoints, szTranslation, szRotation, separation, depth);
	//gpuErrorCheck(hipPeekAtLastError());
	//gpuErrorCheck(hipDeviceSynchronize());
	////TODO: Change so that all data is copied at once instead of point per point.
	//for (int i = 0; i < totalPoints; i++) {
	//	int pos = i * 3;
	//	workspaceModel.addPoint(rawPoints[pos], rawPoints[pos + 1], rawPoints[pos + 2]);
	//}
	//gpuErrorCheck(hipFree(rawPoints));
}

void computeMK3Workspace_cone(ModelData& workspaceModel, float separation, 
	float dmin, float dmax, float range, int insertions, int rotations) 
{
	//if (insertions > 1024) {
	//	insertions = 1024;
	//}
	//if (rotations > 1024) {
	//	rotations = 1024;
	//}

	//float szInsertions = (dmax - dmin) / (float)insertions;
	//float szRotations = 6.28318f / (float)rotations;
	//int totalPoints = insertions * rotations;

	//float *rawPoints = 0;
	//gpuErrorCheck(hipMallocManaged(&rawPoints, totalPoints * workspaceModel.getPointSize()));
	//gpuErrorCheck(hipMemset(rawPoints, 0, totalPoints * workspaceModel.getPointSize()));
	//computeMK3Workspace_cone_cuda<<<insertions, rotations>>>(rawPoints, szInsertions, szRotations, separation, range, dmin, dmax);
	//gpuErrorCheck(hipPeekAtLastError());
	//gpuErrorCheck(hipDeviceSynchronize());
	////TODO: Change so that all data is copied at once instead of point per point.
	//for (int i = 0; i < totalPoints; i++) {
	//	int pos = i * 3;
	//	workspaceModel.addPoint(rawPoints[pos], rawPoints[pos + 1], rawPoints[pos + 2]);
	//}
	//gpuErrorCheck(hipFree(rawPoints));
}

#endif