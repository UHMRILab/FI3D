#include "hip/hip_runtime.h"
#include <fi3d/CUDA/cuda_computations.cuh>

#include <math.h>
#include <iostream>

__global__ void fi3d::CUDA::computeMK3Workspace_sphere_cuda(float* points, float circleStepSize, float pointStepSize, 
	float separation, float depthMin) 
{
	int i = blockIdx.x;
	int j = threadIdx.x;

	float cRange = (i+1) * circleStepSize;
	float cTheta = (j+1) * pointStepSize;

	float x = cRange * cosf(cTheta);
	float y = cRange * sinf(cTheta);
	
	float magnitude = sqrtf(x * x + y * y + separation * separation);

	float px = x + depthMin * x / magnitude;
	float py = y + depthMin * y / magnitude;
	float pz = separation + depthMin * separation / magnitude;

	int position = (i * blockDim.x + j) * 3;
	points[position] = px;
	points[position + 1] = py;
	points[position + 2] = pz;
}

__global__ void fi3d::CUDA::computeMK3Workspace_cone_cuda(float * points, float insertionStepSIze, float pointStepSize,
	float separation, float range, float depthMin, float depthMax) 
{
	int i = blockIdx.x;
	int j = threadIdx.x;

	float cDepth = (i + 1) * insertionStepSIze + depthMin;
	float cTheta = (j + 1) * pointStepSize;

	float x = range * cosf(cTheta);
	float y = range * sinf(cTheta);

	float magnitude = sqrtf(x * x + y * y + separation * separation);

	float px = x + cDepth * x / magnitude;
	float py = y + cDepth * y / magnitude;
	float pz = separation + cDepth * separation / magnitude;

	int pos = (i * blockDim.x + j) * 3;
	points[pos] = px;
	points[pos + 1] = py;
	points[pos + 2] = pz;
}
